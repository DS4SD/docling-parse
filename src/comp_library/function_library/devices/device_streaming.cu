//-*-C++-*-

#ifndef DEVICE_STREAMING_CU
#define DEVICE_STREAMING_CU

namespace func_lib
{
  namespace device_management
  {

    /******************************************
     ***   IMPLEMENTATION STREAM-HANDLES    ***
     ******************************************/
    
    hipStream_t& get_stream_handle(int thread_id, int stream_id)
    {
      assert(thread_id>-1 and thread_id<64);
      assert(stream_id>-1 and stream_id<8);

      static hipStream_t streams[64][8];
      return streams[thread_id][stream_id];

      /*
      switch(thread_id)
        {
        case 0 :
          {
            static cudaStream_t stream[8];
            return stream[stream_id];
          }

        case 1 :
          {
            static cudaStream_t stream[8];
            return stream[stream_id];
          }

        case 2 :
          {
            static cudaStream_t stream[8];
            return stream[stream_id];
          }

        case 3 :
          {
            static cudaStream_t stream[8];
            return stream[stream_id];
          }

        case 4 :
          {
            static cudaStream_t stream[8];
            return stream[stream_id];
          }

        case 5 :
          {
            static cudaStream_t stream[8];
            return stream[stream_id];
          }

        case 6 :
          {
            static cudaStream_t stream[8];
            return stream[stream_id];
          }

        case 7 :
          {
            static cudaStream_t stream[8];
            return stream[stream_id];
          }

        case 8 :
          {
            static cudaStream_t stream[8];
            return stream[stream_id];
          }

        case 9 :
          {
            static cudaStream_t stream[8];
            return stream[stream_id];
          }

        case 10 :
          {
            static cudaStream_t stream[8];
            return stream[stream_id];
          }

        case 11 :
          {
            static cudaStream_t stream[8];
            return stream[stream_id];
          }

        case 12 :
          {
            static cudaStream_t stream[8];
            return stream[stream_id];
          }

        case 13 :
          {
            static cudaStream_t stream[8];
            return stream[stream_id];
          }

        case 14 :
          {
            static cudaStream_t stream[8];
            return stream[stream_id];
          }

        case 15 :
          {
            static cudaStream_t stream[8];
            return stream[stream_id];
          }

        default:
          throw std::logic_error(__FUNCTION__);
        }
      */
    }

    void create_stream_handle(int thread_id)
    {
      for(int stream_id=0; stream_id<8; ++stream_id)
        hipStreamCreate(&get_stream_handle(thread_id, stream_id));
    }

    void destroy_stream_handle(int thread_id)
    {
      for(int stream_id=0; stream_id<8; ++stream_id)
        hipStreamDestroy(get_stream_handle(thread_id, stream_id));
    }

    void synchronize_stream_handle(int thread_id, int stream_id)
    {
      hipStreamSynchronize(get_stream_handle(thread_id, stream_id));
    }

  }

}

#endif
