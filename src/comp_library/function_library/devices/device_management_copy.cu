//-*-C++-*-

#ifndef DEVICE_MANAGEMENT_COPY_CU
#define DEVICE_MANAGEMENT_COPY_CU

namespace func_lib
{
  namespace device_management
  {
    namespace gpu_memory_management
    {
      template<typename scalar_type, typename index_type>
      void memcpy(std::string kind, scalar_type* target_ptr, scalar_type* source_ptr, index_type N)
      {
        //cout << kind /*<< "\t" << get_cudaMemcpyKind(kind)*/ << "\t" << target_ptr << "\t" << source_ptr << "\t" << N << "\n";

        hipError_t error;

        if(kind=="host_to_host")
          {
            error = hipMemcpy(target_ptr, source_ptr, sizeof(scalar_type)*N, hipMemcpyHostToHost);
            assert(check_for_cuda_error(error, __FUNCTION__, __FILE__, __LINE__));
          }

        if(kind=="host_to_device")
          {	    
            error = hipMemcpy(target_ptr, source_ptr, sizeof(scalar_type)*N, hipMemcpyHostToDevice);
            assert(check_for_cuda_error(error, __FUNCTION__, __FILE__, __LINE__));
          }

        if(kind=="device_to_host")
          {
            error = hipMemcpy(target_ptr, source_ptr, sizeof(scalar_type)*N, hipMemcpyDeviceToHost);
            assert(check_for_cuda_error(error, __FUNCTION__, __FILE__, __LINE__));
          }

        if(kind=="device_to_device")
          {
            error = hipMemcpy(target_ptr, source_ptr, sizeof(scalar_type)*N, hipMemcpyDeviceToDevice);
            assert(check_for_cuda_error(error, __FUNCTION__, __FILE__, __LINE__));
          }

        assert(check_for_cuda_error(error, __FUNCTION__, __FILE__, __LINE__));
      }

      template<typename scalar_type, typename index_type>
      void memcpy_asynchronous(std::string kind, scalar_type* target_ptr, scalar_type* source_ptr, index_type N, 
			       int thread_id, int stream_id)
      {
	hipError_t error = hipErrorUnknown;

	hipStream_t& stream = get_stream_handle(thread_id, stream_id);

        if(kind=="host_to_host")
          {
            error = hipMemcpyAsync(target_ptr, source_ptr, sizeof(scalar_type)*N, hipMemcpyHostToHost, stream);
            assert(check_for_cuda_error(error, __FUNCTION__, __FILE__, __LINE__));
          }

        if(kind=="host_to_device")
          {	    
            error = hipMemcpyAsync(target_ptr, source_ptr, sizeof(scalar_type)*N, hipMemcpyHostToDevice, stream);
            assert(check_for_cuda_error(error, __FUNCTION__, __FILE__, __LINE__));
          }

        if(kind=="device_to_host")
          {
            error = hipMemcpyAsync(target_ptr, source_ptr, sizeof(scalar_type)*N, hipMemcpyDeviceToHost, stream);
            assert(check_for_cuda_error(error, __FUNCTION__, __FILE__, __LINE__));
          }

        if(kind=="device_to_device")
          {
            error = hipMemcpyAsync(target_ptr, source_ptr, sizeof(scalar_type)*N, hipMemcpyDeviceToDevice, stream);
            assert(check_for_cuda_error(error, __FUNCTION__, __FILE__, __LINE__));
          }

	assert(check_for_cuda_error(error, __FUNCTION__, __FILE__, __LINE__));
      }

      template<typename scalar_type, typename index_type>
      void memcpy_2D(std::string kind,
		     scalar_type* target_ptr, std::pair<index_type, index_type> target_size, std::pair<index_type, index_type> target_pad,
                     scalar_type* source_ptr, std::pair<index_type, index_type> source_size, std::pair<index_type, index_type> source_pad)
      {
	/*
        cout << "\n" << kind << " : " 
	     << "\t(" << source_size.first << ", " << source_size.second << ") in " 
	     << "  [" << source_pad.first << ", " << source_pad.second << "] ---> "
	     << "\t(" << target_size.first << ", " << target_size.second << ") in "
	     << "  [" << target_pad.first << ", " << target_pad.second << "] \n\n";
	*/

        hipError_t error;

        if(kind=="host_to_host")
          {
            error = hipMemcpy2D(target_ptr, target_pad.first*sizeof(scalar_type), 
				 source_ptr, source_pad.first*sizeof(scalar_type), 
				 target_size.first*sizeof(scalar_type), /*sizeof(scalar_type)*/target_size.second, 
				 hipMemcpyHostToHost);
            assert(check_for_cuda_error(error, __FUNCTION__, __FILE__, __LINE__));
          }

        if(kind=="host_to_device")
          {
	    //cout << source_ptr[0] << " --> "; 
            error = hipMemcpy2D(target_ptr, target_pad.first*sizeof(scalar_type), 
				 source_ptr, source_pad.first*sizeof(scalar_type), 
				 target_size.first*sizeof(scalar_type), /*sizeof(scalar_type)*/target_size.second, 
				 //target_size.second, sizeof(scalar_type)*target_size.first, 
				 hipMemcpyHostToDevice);
            assert(check_for_cuda_error(error, __FUNCTION__, __FILE__, __LINE__));
          }

        if(kind=="device_to_host")
          {
	    //cout << target_ptr[0] << "\n"; 
            error = hipMemcpy2D(target_ptr, target_pad.first*sizeof(scalar_type), 
				 source_ptr, source_pad.first*sizeof(scalar_type), 
				 target_size.first*sizeof(scalar_type), /*sizeof(scalar_type)*/target_size.second, 
				 //target_size.second, sizeof(scalar_type)*target_size.first, 
				 hipMemcpyDeviceToHost);
            assert(check_for_cuda_error(error, __FUNCTION__, __FILE__, __LINE__));
          }

        if(kind=="device_to_device")
          {
            error = hipMemcpy2D(target_ptr, target_pad.first*sizeof(scalar_type), 
				 source_ptr, source_pad.first*sizeof(scalar_type), 
				 target_size.first*sizeof(scalar_type), /*sizeof(scalar_type)*/target_size.second, 
				 hipMemcpyDeviceToDevice);
            assert(check_for_cuda_error(error, __FUNCTION__, __FILE__, __LINE__));
          }

        assert(check_for_cuda_error(error, __FUNCTION__, __FILE__, __LINE__));
      }

      template<typename scalar_type, typename index_type>
      void memcpy_2D_asynchronous(std::string kind,
				  scalar_type* target_ptr, std::pair<index_type, index_type> target_size, std::pair<index_type, index_type> target_pad,
				  scalar_type* source_ptr, std::pair<index_type, index_type> source_size, std::pair<index_type, index_type> source_pad,
				  int thread_id, int stream_id)
      {
	/*
        cout << "\n" << kind << "-asynchronous : " 
	     << "\t(" << source_size.first << ", " << source_size.second << ") in " 
	     << "  [" << source_pad.first << ", " << source_pad.second << "] ---> "
	     << "\t(" << target_size.first << ", " << target_size.second << ") in "
	     << "  [" << target_pad.first << ", " << target_pad.second << "] \n\n";
	*/

	hipError_t error = hipErrorUnknown;

	hipStream_t& stream = get_stream_handle(thread_id, stream_id);

        if(kind=="host_to_host")
	  {
            error = hipMemcpy2DAsync(target_ptr, target_pad.first*sizeof(scalar_type), 
				      source_ptr, source_pad.first*sizeof(scalar_type), 
				      target_size.first*sizeof(scalar_type), target_size.second, 
				      hipMemcpyHostToHost,
				      stream);
            assert(check_for_cuda_error(error, __FUNCTION__, __FILE__, __LINE__));
	  }

        if(kind=="host_to_device")
	  {
            error = hipMemcpy2DAsync(target_ptr, target_pad.first*sizeof(scalar_type), 
				      source_ptr, source_pad.first*sizeof(scalar_type), 
				      target_size.first*sizeof(scalar_type), target_size.second, 
				      hipMemcpyHostToDevice,
				      stream);
            assert(check_for_cuda_error(error, __FUNCTION__, __FILE__, __LINE__));
	  }

        if(kind=="device_to_host")
	  {
            error = hipMemcpy2DAsync(target_ptr, target_pad.first*sizeof(scalar_type), 
				      source_ptr, source_pad.first*sizeof(scalar_type), 
				      target_size.first*sizeof(scalar_type), target_size.second, 
				      hipMemcpyDeviceToHost,
				      stream);
            assert(check_for_cuda_error(error, __FUNCTION__, __FILE__, __LINE__));
	  }

        if(kind=="device_to_device")
	  {
            error = hipMemcpy2DAsync(target_ptr, target_pad.first*sizeof(scalar_type), 
				      source_ptr, source_pad.first*sizeof(scalar_type), 
				      target_size.first*sizeof(scalar_type), target_size.second, 
				      hipMemcpyDeviceToDevice,
				      stream);
            assert(check_for_cuda_error(error, __FUNCTION__, __FILE__, __LINE__));
	  }
	
	assert(check_for_cuda_error(error, __FUNCTION__, __FILE__, __LINE__));
      }

      // template instantiations

      /*****************************
       *** 
       ***  memcpy
       ***
       *****************************/

      // int
      template void memcpy<short , int>(std::string kind, short*  target_ptr, short*  source_ptr, int N);
      template void memcpy<int   , int>(std::string kind, int*    target_ptr, int*    source_ptr, int N);
      template void memcpy<long  , int>(std::string kind, long*   target_ptr, long*   source_ptr, int N);

      template void memcpy<unsigned short , int>(std::string kind, unsigned short*  target_ptr, unsigned short*  source_ptr, int N);
      template void memcpy<unsigned int   , int>(std::string kind, unsigned int*    target_ptr, unsigned int*    source_ptr, int N);
      template void memcpy<unsigned long  , int>(std::string kind, unsigned long*   target_ptr, unsigned long*   source_ptr, int N);

      template void memcpy<float , int>(std::string kind, float*  target_ptr, float*  source_ptr, int N);
      template void memcpy<double, int>(std::string kind, double* target_ptr, double* source_ptr, int N);

      template void memcpy<std::complex<float> , int>(std::string kind, std::complex<float>*  target_ptr, std::complex<float>*  source_ptr, int N);
      template void memcpy<std::complex<double>, int>(std::string kind, std::complex<double>* target_ptr, std::complex<double>* source_ptr, int N);

      // unsigned int
      template void memcpy<short , unsigned int>(std::string kind, short*  target_ptr, short*  source_ptr, unsigned int N);
      template void memcpy<int   , unsigned int>(std::string kind, int*    target_ptr, int*    source_ptr, unsigned int N);
      template void memcpy<long  , unsigned int>(std::string kind, long*   target_ptr, long*   source_ptr, unsigned int N);

      template void memcpy<unsigned short , unsigned int>(std::string kind, unsigned short*  target_ptr, unsigned short*  source_ptr, unsigned int N);
      template void memcpy<unsigned int   , unsigned int>(std::string kind, unsigned int*    target_ptr, unsigned int*    source_ptr, unsigned int N);
      template void memcpy<unsigned long  , unsigned int>(std::string kind, unsigned long*   target_ptr, unsigned long*   source_ptr, unsigned int N);

      template void memcpy<float , unsigned int>(std::string kind, float*  target_ptr, float*  source_ptr, unsigned int N);
      template void memcpy<double, unsigned int>(std::string kind, double* target_ptr, double* source_ptr, unsigned int N);

      template void memcpy<std::complex<float> , unsigned int>(std::string kind, std::complex<float>*  target_ptr, std::complex<float>*  source_ptr, unsigned int N);
      template void memcpy<std::complex<double>, unsigned int>(std::string kind, std::complex<double>* target_ptr, std::complex<double>* source_ptr, unsigned int N);

      // unsigned long
      template void memcpy<short , unsigned long>(std::string kind, short*  target_ptr, short*  source_ptr, unsigned long N);
      template void memcpy<int   , unsigned long>(std::string kind, int*    target_ptr, int*    source_ptr, unsigned long N);
      template void memcpy<long  , unsigned long>(std::string kind, long*   target_ptr, long*   source_ptr, unsigned long N);

      template void memcpy<unsigned short , unsigned long>(std::string kind, unsigned short*  target_ptr, unsigned short*  source_ptr, unsigned long N);
      template void memcpy<unsigned int   , unsigned long>(std::string kind, unsigned int*    target_ptr, unsigned int*    source_ptr, unsigned long N);
      template void memcpy<unsigned long  , unsigned long>(std::string kind, unsigned long*   target_ptr, unsigned long*   source_ptr, unsigned long N);

      template void memcpy<float , unsigned long>(std::string kind, float*  target_ptr, float*  source_ptr, unsigned long N);
      template void memcpy<double, unsigned long>(std::string kind, double* target_ptr, double* source_ptr, unsigned long N);

      template void memcpy<std::complex<float> , unsigned long>(std::string kind, std::complex<float>*  target_ptr, std::complex<float>*  source_ptr, unsigned long N);
      template void memcpy<std::complex<double>, unsigned long>(std::string kind, std::complex<double>* target_ptr, std::complex<double>* source_ptr, unsigned long N);

      // unsigned long long
      template void memcpy<short , unsigned long long>(std::string kind, short*  target_ptr, short*  source_ptr, unsigned long long N);
      template void memcpy<int   , unsigned long long>(std::string kind, int*    target_ptr, int*    source_ptr, unsigned long long N);
      template void memcpy<long  , unsigned long long>(std::string kind, long*   target_ptr, long*   source_ptr, unsigned long long N);

      template void memcpy<unsigned short , unsigned long long>(std::string kind, unsigned short*  target_ptr, unsigned short*  source_ptr, unsigned long long N);
      template void memcpy<unsigned int   , unsigned long long>(std::string kind, unsigned int*    target_ptr, unsigned int*    source_ptr, unsigned long long N);
      template void memcpy<unsigned long  , unsigned long long>(std::string kind, unsigned long*   target_ptr, unsigned long*   source_ptr, unsigned long long N);

      template void memcpy<float , unsigned long long>(std::string kind, float*  target_ptr, float*  source_ptr, unsigned long long N);
      template void memcpy<double, unsigned long long>(std::string kind, double* target_ptr, double* source_ptr, unsigned long long N);

      template void memcpy<std::complex<float> , unsigned long long>(std::string kind, std::complex<float>*  target_ptr, std::complex<float>*  source_ptr, unsigned long long N);
      template void memcpy<std::complex<double>, unsigned long long>(std::string kind, std::complex<double>* target_ptr, std::complex<double>* source_ptr, unsigned long long N);

      /*****************************
       *** 
       ***  memcpy-asynchronous
       ***
       *****************************/

      // unsigned long
      template void memcpy_asynchronous<int   , unsigned long>(std::string kind, int*  target_ptr, int*  source_ptr, unsigned long N, 
							       int thread_id, int stream_id);

      template void memcpy_asynchronous<float , unsigned long>(std::string kind, float*  target_ptr, float*  source_ptr, unsigned long N,
							       int thread_id, int stream_id);
      template void memcpy_asynchronous<double, unsigned long>(std::string kind, double* target_ptr, double* source_ptr, unsigned long N, 
							       int thread_id, int stream_id);

      template void memcpy_asynchronous<std::complex<float> , unsigned long>(std::string kind, std::complex<float>*  target_ptr, std::complex<float>*  source_ptr, unsigned long N, 
									     int thread_id, int stream_id);
      template void memcpy_asynchronous<std::complex<double>, unsigned long>(std::string kind, std::complex<double>* target_ptr, std::complex<double>* source_ptr, unsigned long N, 
									     int thread_id, int stream_id);

      /*****************************
       *** 
       ***  memcpy-2D
       ***
       *****************************/
      template void memcpy_2D<short, int>(std::string kind,
					short* target_ptr, std::pair<int, int> target_step, std::pair<int, int> target_size,
					short* source_ptr, std::pair<int, int> source_step, std::pair<int, int> source_size);

      template void memcpy_2D<unsigned short, int>(std::string kind,
					unsigned short* target_ptr, std::pair<int, int> target_step, std::pair<int, int> target_size,
					unsigned short* source_ptr, std::pair<int, int> source_step, std::pair<int, int> source_size);

      template void memcpy_2D<int, int>(std::string kind,
					int* target_ptr, std::pair<int, int> target_step, std::pair<int, int> target_size,
					int* source_ptr, std::pair<int, int> source_step, std::pair<int, int> source_size);

      template void memcpy_2D<unsigned int, int>(std::string kind,
					unsigned int* target_ptr, std::pair<int, int> target_step, std::pair<int, int> target_size,
					unsigned int* source_ptr, std::pair<int, int> source_step, std::pair<int, int> source_size);

      template void memcpy_2D<long, int>(std::string kind,
					long* target_ptr, std::pair<int, int> target_step, std::pair<int, int> target_size,
					long* source_ptr, std::pair<int, int> source_step, std::pair<int, int> source_size);

      template void memcpy_2D<unsigned long, int>(std::string kind,
						  unsigned long* target_ptr, std::pair<int, int> target_step, std::pair<int, int> target_size,
						  unsigned long* source_ptr, std::pair<int, int> source_step, std::pair<int, int> source_size);
      
      template void memcpy_2D<float, int>(std::string kind,
					  float* target_ptr, std::pair<int, int> target_step, std::pair<int, int> target_size,
					  float* source_ptr, std::pair<int, int> source_step, std::pair<int, int> source_size);

      template void memcpy_2D<double, int>(std::string kind,
					   double* target_ptr, std::pair<int, int> target_step, std::pair<int, int> target_size,
					   double* source_ptr, std::pair<int, int> source_step, std::pair<int, int> source_size);

      template void memcpy_2D<std::complex<float>, int>(std::string kind,
							std::complex<float>* target_ptr, std::pair<int, int> target_step, std::pair<int, int> target_size,
							std::complex<float>* source_ptr, std::pair<int, int> source_step, std::pair<int, int> source_size);

      template void memcpy_2D<std::complex<double>, int>(std::string kind,
							 std::complex<double>* target_ptr, std::pair<int, int> target_step, std::pair<int, int> target_size,
							 std::complex<double>* source_ptr, std::pair<int, int> source_step, std::pair<int, int> source_size);


      /*****************************
       *** 
       ***  memcpy-2D-asynchronous
       ***
       *****************************/

      template void memcpy_2D_asynchronous<int, int>(std::string kind,
						     int* target_ptr, std::pair<int, int> target_step, std::pair<int, int> target_size,
						     int* source_ptr, std::pair<int, int> source_step, std::pair<int, int> source_size,
						     int thread_id, int stream_id);
      
      template void memcpy_2D_asynchronous<float, int>(std::string kind,
						       float* target_ptr, std::pair<int, int> target_step, std::pair<int, int> target_size,
						       float* source_ptr, std::pair<int, int> source_step, std::pair<int, int> source_size,
						       int thread_id, int stream_id);
      
      template void memcpy_2D_asynchronous<double, int>(std::string kind,
							double* target_ptr, std::pair<int, int> target_step, std::pair<int, int> target_size,
							double* source_ptr, std::pair<int, int> source_step, std::pair<int, int> source_size,
							int thread_id, int stream_id);
      
      template void memcpy_2D_asynchronous<std::complex<float>, int>(std::string kind,
								     std::complex<float>* target_ptr, std::pair<int, int> target_step, std::pair<int, int> target_size,
								     std::complex<float>* source_ptr, std::pair<int, int> source_step, std::pair<int, int> source_size,
								     int thread_id, int stream_id);
      
      template void memcpy_2D_asynchronous<std::complex<double>, int>(std::string kind,
								      std::complex<double>* target_ptr, std::pair<int, int> target_step, std::pair<int, int> target_size,
								      std::complex<double>* source_ptr, std::pair<int, int> source_step, std::pair<int, int> source_size,
								      int thread_id, int stream_id);

    }

  }

}

#endif
