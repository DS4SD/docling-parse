//-*-C++-*-

#ifndef DEVICE_MANAGEMENT_GPU_CU
#define DEVICE_MANAGEMENT_GPU_CU

namespace func_lib
{
  namespace device_management
  {
    namespace gpu_memory_management
    {
      bool check_for_cuda_error(hipError_t error, const char* function, const char* file, const size_t line)
      {
        if(error != hipSuccess)
          {
            std::stringstream ss;
            ss << "\n\nerror (" << hipGetErrorString(error) << ") detected  \n";
            ss << "\t function :" << function << "\n";
            ss << "\t file     :" << file << "\n";
            ss << "\t line     :" << line << "\n";

	    std::cout << ss.str();

            throw std::logic_error(__FUNCTION__);
          }

        return true;
      }

      template<typename scalar_type, typename index_type>
      void malloc(scalar_type*& ptr, index_type N)
      {
        hipError_t error = hipMalloc(&ptr, sizeof(scalar_type)*N);
        assert(check_for_cuda_error(error, __FUNCTION__, __FILE__, __LINE__));
      }

      template<typename scalar_type, typename index_type>
      void malloc_pinned_host_memory(scalar_type*& ptr, index_type N)
      {
        hipError_t error = hipHostMalloc(&ptr, sizeof(scalar_type)*N);
        assert(check_for_cuda_error(error, __FUNCTION__, __FILE__, __LINE__));
      }

      template<typename scalar_type>
      void free(scalar_type*& ptr)
      {
        hipError_t error = hipFree(ptr);
        assert(check_for_cuda_error(error, __FUNCTION__, __FILE__, __LINE__));
      }

      template<typename scalar_type>
      void free_pinned_host_memory(scalar_type*& ptr)
      {
        hipError_t error = hipHostFree(ptr);
	assert(check_for_cuda_error(error, __FUNCTION__, __FILE__, __LINE__));
      }

      template<typename scalar_type, typename index_type>
      void register_host_memory(scalar_type*& ptr, index_type N)
      {
	hipError_t error = hipHostRegister(ptr, sizeof(scalar_type)*N, hipHostRegisterPortable);
	assert(check_for_cuda_error(error, __FUNCTION__, __FILE__, __LINE__));
      }

      template<typename scalar_type>
      void unregister_host_memory(scalar_type*& ptr)
      {	
	hipError_t error = hipHostUnregister(ptr);
	assert(check_for_cuda_error(error, __FUNCTION__, __FILE__, __LINE__));
      }

      template<typename scalar_type, typename index_type>
      void memset(scalar_type*& ptr, scalar_type value, index_type N)
      {
        hipError_t error = hipMemset(ptr, value, sizeof(scalar_type)*N);
        assert(check_for_cuda_error(error, __FUNCTION__, __FILE__, __LINE__));
      }

      // template instantiations

      // int 
      template void malloc<         short, int>(         short*& ptr , int N);
      template void malloc<unsigned short, int>(unsigned short*& ptr , int N);

      template void malloc<         int, int>(         int*& ptr , int N);
      template void malloc<unsigned int, int>(unsigned int*& ptr , int N);

      template void malloc<bool  , int>(bool  *& ptr , int N);
      template void malloc<float , int>(float *& ptr , int N);
      template void malloc<double, int>(double*& ptr, int N);

      template void malloc<std::complex<float> , int>(std::complex<float>*& ptr , int N);
      template void malloc<std::complex<double>, int>(std::complex<double>*& ptr, int N);

      template void malloc_pinned_host_memory<         int, int>(         int*& ptr , int N);
      template void malloc_pinned_host_memory<unsigned int, int>(unsigned int*& ptr , int N);

      template void malloc_pinned_host_memory<bool  , int>(bool  *& ptr , int N);
      template void malloc_pinned_host_memory<float , int>(float *& ptr , int N);
      template void malloc_pinned_host_memory<double, int>(double*& ptr, int N);

      template void malloc_pinned_host_memory<std::complex<float> , int>(std::complex<float>*& ptr , int N);
      template void malloc_pinned_host_memory<std::complex<double>, int>(std::complex<double>*& ptr, int N);

      // unsigned int 
      template void malloc<         short, unsigned int>(         short*& ptr , unsigned int N);
      template void malloc<unsigned short, unsigned int>(unsigned short*& ptr , unsigned int N);

      template void malloc<         int, unsigned int>(         int*& ptr , unsigned int N);
      template void malloc<unsigned int, unsigned int>(unsigned int*& ptr , unsigned int N);

      template void malloc<float , unsigned int>(float*& ptr , unsigned int N);
      template void malloc<double, unsigned int>(double*& ptr, unsigned int N);

      template void malloc<std::complex<float> , unsigned int>(std::complex<float>*& ptr , unsigned int N);
      template void malloc<std::complex<double>, unsigned int>(std::complex<double>*& ptr, unsigned int N);

      template void malloc_pinned_host_memory<         int, unsigned int>(         int*& ptr , unsigned int N);
      template void malloc_pinned_host_memory<unsigned int, unsigned int>(unsigned int*& ptr , unsigned int N);

      template void malloc_pinned_host_memory<float , unsigned int>(float*& ptr , unsigned int N);
      template void malloc_pinned_host_memory<double, unsigned int>(double*& ptr, unsigned int N);

      template void malloc_pinned_host_memory<std::complex<float> , unsigned int>(std::complex<float>*& ptr , unsigned int N);
      template void malloc_pinned_host_memory<std::complex<double>, unsigned int>(std::complex<double>*& ptr, unsigned int N);

      // unsigned long 
      template void malloc<         short, unsigned long>(         short*& ptr , unsigned long N);
      template void malloc<unsigned short, unsigned long>(unsigned short*& ptr , unsigned long N);

      template void malloc<int          , unsigned long>(         int *& ptr , unsigned long N);
      template void malloc<unsigned int , unsigned long>(unsigned int *& ptr , unsigned long N);

      template void malloc<long         , unsigned long>(         long *& ptr , unsigned long N);
      template void malloc<unsigned long, unsigned long>(unsigned long *& ptr , unsigned long N);
      
      template void malloc<bool, unsigned long>(bool*& ptr , unsigned long N);

      template void malloc<float , unsigned long>(float*& ptr , unsigned long N);
      template void malloc<double, unsigned long>(double*& ptr, unsigned long N);

      template void malloc<std::complex<float> , unsigned long>(std::complex<float>*& ptr , unsigned long N);
      template void malloc<std::complex<double>, unsigned long>(std::complex<double>*& ptr, unsigned long N);

      template void malloc_pinned_host_memory<int , unsigned long>(int *& ptr , unsigned long N);
      template void malloc_pinned_host_memory<bool, unsigned long>(bool*& ptr , unsigned long N);

      template void malloc_pinned_host_memory<float , unsigned long>(float*& ptr , unsigned long N);
      template void malloc_pinned_host_memory<double, unsigned long>(double*& ptr, unsigned long N);

      template void malloc_pinned_host_memory<std::complex<float> , unsigned long>(std::complex<float>*& ptr , unsigned long N);
      template void malloc_pinned_host_memory<std::complex<double>, unsigned long>(std::complex<double>*& ptr, unsigned long N);

      // unsigned long long
      template void malloc<         short, unsigned long long>(         short*& ptr , unsigned long long N);
      template void malloc<unsigned short, unsigned long long>(unsigned short*& ptr , unsigned long long N);

      template void malloc<int , unsigned long long>(int *& ptr , unsigned long long N);
      template void malloc<bool, unsigned long long>(bool*& ptr , unsigned long long N);
      
      template void malloc<float , unsigned long long>(float*& ptr , unsigned long long N);
      template void malloc<double, unsigned long long>(double*& ptr, unsigned long long N);

      template void malloc<std::complex<float> , unsigned long long>(std::complex<float>*& ptr , unsigned long long N);
      template void malloc<std::complex<double>, unsigned long long>(std::complex<double>*& ptr, unsigned long long N);

      template void malloc_pinned_host_memory<int , unsigned long long>(int *& ptr , unsigned long long N);
      template void malloc_pinned_host_memory<bool, unsigned long long>(bool*& ptr , unsigned long long N);
      
      template void malloc_pinned_host_memory<float , unsigned long long>(float*& ptr , unsigned long long N);
      template void malloc_pinned_host_memory<double, unsigned long long>(double*& ptr, unsigned long long N);

      template void malloc_pinned_host_memory<std::complex<float> , unsigned long long>(std::complex<float>*& ptr , unsigned long long N);
      template void malloc_pinned_host_memory<std::complex<double>, unsigned long long>(std::complex<double>*& ptr, unsigned long long N);

      // template instantiations
      template void free<char     >(char      *& ptr );
      template void free<short    >(short     *& ptr );
      template void free<int      >(int       *& ptr );
      template void free<long     >(long      *& ptr );
      template void free<long long>(long long *& ptr );
      
      template void free<unsigned short    >(unsigned short    *& ptr );
      template void free<unsigned int      >(unsigned int      *& ptr );
      template void free<unsigned long     >(unsigned long     *& ptr );
      template void free<unsigned long long>(unsigned long long*& ptr );
      
      template void free<bool  >(bool  *& ptr );
      template void free<float >(float *& ptr );
      template void free<double>(double*& ptr);

      template void free<std::complex<float>  >(std::complex<float>*& ptr );
      template void free<std::complex<double> >(std::complex<double>*& ptr);

      template void free_pinned_host_memory<int   >(int   *& ptr );
      template void free_pinned_host_memory<bool  >(bool  *& ptr );
      template void free_pinned_host_memory<float >(float *& ptr );
      template void free_pinned_host_memory<double>(double*& ptr);

      template void free_pinned_host_memory<std::complex<float>  >(std::complex<float>*& ptr );
      template void free_pinned_host_memory<std::complex<double> >(std::complex<double>*& ptr);
      
      // template instantiations
      template void memset<             int    , int>(               int*&   ptr,                int   value, int N);
      template void memset<             float  , int>(             float*&   ptr,              float   value, int N);
      template void memset<             double , int>(             double*&  ptr,              double  value, int N);

      //template void memset<std::complex<float> , int>(std::complex<float >*& ptr, std::complex<float>  value, int N);
      //template void memset<std::complex<double>, int>(std::complex<double>*& ptr, std::complex<double> value, int N);

      /*
      template void memset<             float  , long>(             float*&   ptr,              float   value, long N);
      template void memset<             double , long>(             double*&  ptr,              double  value, long N);
      template void memset<std::complex<float> , long>(std::complex<float >*& ptr, std::complex<float>  value, long N);
      template void memset<std::complex<double>, long>(std::complex<double>*& ptr, std::complex<double> value, long N);

      template void memset<             float  , unsigned int>(             float*&   ptr,              float   value, unsigned int N);
      template void memset<             double , unsigned int>(             double*&  ptr,              double  value, unsigned int N);
      template void memset<std::complex<float> , unsigned int>(std::complex<float >*& ptr, std::complex<float>  value, unsigned int N);
      template void memset<std::complex<double>, unsigned int>(std::complex<double>*& ptr, std::complex<double> value, unsigned int N);
      */

      template void memset<               int  , unsigned long>(               int*&   ptr,                int   value, unsigned long N);
      template void memset<             float  , unsigned long>(             float*&   ptr,              float   value, unsigned long N);
      template void memset<             double , unsigned long>(             double*&  ptr,              double  value, unsigned long N);
      //template void memset<std::complex<float> , unsigned long>(std::complex<float >*& ptr, std::complex<float>  value, unsigned long N);
      //template void memset<std::complex<double>, unsigned long>(std::complex<double>*& ptr, std::complex<double> value, unsigned long N);

      /*
      template void memset<             float  , unsigned long long>(             float*&   ptr,              float   value, unsigned long long N);
      template void memset<             double , unsigned long long>(             double*&  ptr,              double  value, unsigned long long N);
      template void memset<std::complex<float> , unsigned long long>(std::complex<float >*& ptr, std::complex<float>  value, unsigned long long N);
      template void memset<std::complex<double>, unsigned long long>(std::complex<double>*& ptr, std::complex<double> value, unsigned long long N);
      */


      // template instantiations
      template void register_host_memory<short               , short>(short               *& ptr,  short N);
      template void register_host_memory<unsigned short      , short>(unsigned short      *& ptr,  short N);
      template void register_host_memory<int                 , short>(int                 *& ptr,  short N);
      template void register_host_memory<unsigned int        , short>(unsigned int        *& ptr,  short N);
      template void register_host_memory<long                , short>(long                *& ptr,  short N);
      template void register_host_memory<unsigned long       , short>(unsigned long       *& ptr,  short N);
      template void register_host_memory<long long           , short>(long long           *& ptr,  short N);
      template void register_host_memory<unsigned long long  , short>(unsigned long long  *& ptr,  short N);
      template void register_host_memory<bool                , short>(bool                *& ptr,  short N);
      template void register_host_memory<float               , short>(float               *& ptr,  short N);
      template void register_host_memory<double              , short>(double              *& ptr,  short N);
      template void register_host_memory<std::complex<float> , short>(std::complex<float> *& ptr,  short N);
      template void register_host_memory<std::complex<double>, short>(std::complex<double>*& ptr,  short N);

      template void register_host_memory<short               , int>(short               *& ptr, int N);
      template void register_host_memory<unsigned short      , int>(unsigned short      *& ptr, int N);
      template void register_host_memory<int                 , int>(int                 *& ptr, int N);
      template void register_host_memory<unsigned int        , int>(unsigned int        *& ptr, int N);
      template void register_host_memory<long                , int>(long                *& ptr, int N);
      template void register_host_memory<unsigned long       , int>(unsigned long       *& ptr, int N);
      template void register_host_memory<long long           , int>(long long           *& ptr, int N);
      template void register_host_memory<unsigned long long  , int>(unsigned long long  *& ptr, int N);
      template void register_host_memory<bool                , int>(bool                *& ptr, int N);
      template void register_host_memory<float               , int>(float               *& ptr, int N);
      template void register_host_memory<double              , int>(double              *& ptr, int N);
      template void register_host_memory<std::complex<float> , int>(std::complex<float> *& ptr, int N);
      template void register_host_memory<std::complex<double>, int>(std::complex<double>*& ptr, int N);

      template void register_host_memory<short               , long>(short               *& ptr,  long N);
      template void register_host_memory<unsigned short      , long>(unsigned short      *& ptr,  long N);
      template void register_host_memory<int                 , long>(int                 *& ptr,  long N);
      template void register_host_memory<unsigned int        , long>(unsigned int        *& ptr,  long N);
      template void register_host_memory<long                , long>(long                *& ptr,  long N);
      template void register_host_memory<unsigned long       , long>(unsigned long       *& ptr,  long N);
      template void register_host_memory<long long           , long>(long long           *& ptr, long N);
      template void register_host_memory<unsigned long long  , long>(unsigned long long  *& ptr, long N);
      template void register_host_memory<bool                , long>(bool                *& ptr, long N);
      template void register_host_memory<float               , long>(float               *& ptr, long N);
      template void register_host_memory<double              , long>(double              *& ptr, long N);
      template void register_host_memory<std::complex<float> , long>(std::complex<float> *& ptr, long N);
      template void register_host_memory<std::complex<double>, long>(std::complex<double>*& ptr, long N);

      template void register_host_memory<short               , long long>(short               *& ptr,   long long N);
      template void register_host_memory<unsigned short      , long long>(unsigned short      *& ptr,   long long N);
      template void register_host_memory<int                 , long long>(int                 *& ptr,   long long N);
      template void register_host_memory<unsigned int        , long long>(unsigned int        *& ptr,   long long N);
      template void register_host_memory<long                , long long>(long                *& ptr,   long long N);
      template void register_host_memory<unsigned long       , long long>(unsigned long       *& ptr,   long long N);
      template void register_host_memory<long long           , long long>(long long           *& ptr, long long N);
      template void register_host_memory<unsigned long long  , long long>(unsigned long long  *& ptr, long long N);
      template void register_host_memory<bool                , long long>(bool                *& ptr, long long N);
      template void register_host_memory<float               , long long>(float               *& ptr, long long N);
      template void register_host_memory<double              , long long>(double              *& ptr, long long N);
      template void register_host_memory<std::complex<float> , long long>(std::complex<float> *& ptr, long long N);
      template void register_host_memory<std::complex<double>, long long>(std::complex<double>*& ptr, long long N);

      // template instantiations
      template void register_host_memory<short               , unsigned short>(short               *& ptr,  unsigned short N);
      template void register_host_memory<unsigned short      , unsigned short>(unsigned short      *& ptr,  unsigned short N);
      template void register_host_memory<int                 , unsigned short>(int                 *& ptr,  unsigned short N);
      template void register_host_memory<unsigned int        , unsigned short>(unsigned int        *& ptr,  unsigned short N);
      template void register_host_memory<long                , unsigned short>(long                *& ptr,  unsigned short N);
      template void register_host_memory<unsigned long       , unsigned short>(unsigned long       *& ptr,  unsigned short N);
      template void register_host_memory<long long           , unsigned short>(long long           *& ptr,  unsigned short N);
      template void register_host_memory<unsigned long long  , unsigned short>(unsigned long long  *& ptr,  unsigned short N);
      template void register_host_memory<bool                , unsigned short>(bool                *& ptr,  unsigned short N);
      template void register_host_memory<float               , unsigned short>(float               *& ptr,  unsigned short N);
      template void register_host_memory<double              , unsigned short>(double              *& ptr,  unsigned short N);
      template void register_host_memory<std::complex<float> , unsigned short>(std::complex<float> *& ptr,  unsigned short N);
      template void register_host_memory<std::complex<double> , unsigned short>(std::complex<double> *& ptr,  unsigned short N);

      template void register_host_memory<short               , unsigned int>(short               *& ptr,   unsigned int N);
      template void register_host_memory<unsigned short      , unsigned int>(unsigned short      *& ptr,   unsigned int N);
      template void register_host_memory<int                 , unsigned int>(int                 *& ptr,   unsigned int N);
      template void register_host_memory<unsigned int        , unsigned int>(unsigned int        *& ptr,   unsigned int N);
      template void register_host_memory<long                , unsigned int>(long                *& ptr,   unsigned int N);
      template void register_host_memory<unsigned long       , unsigned int>(unsigned long       *& ptr,   unsigned int N);
      template void register_host_memory<long long           , unsigned int>(long long           *& ptr, unsigned int N);
      template void register_host_memory<unsigned long long  , unsigned int>(unsigned long long  *& ptr, unsigned int N);
      template void register_host_memory<bool                , unsigned int>(bool                *& ptr, unsigned int N);
      template void register_host_memory<float               , unsigned int>(float               *& ptr, unsigned int N);
      template void register_host_memory<double              , unsigned int>(double              *& ptr, unsigned int N);
      template void register_host_memory<std::complex<float> , unsigned int>(std::complex<float> *& ptr, unsigned int N);
      template void register_host_memory<std::complex<double>, unsigned int>(std::complex<double>*& ptr, unsigned int N);


      template void register_host_memory<short               , unsigned long>(short               *& ptr,   unsigned long N);
      template void register_host_memory<unsigned short      , unsigned long>(unsigned short      *& ptr,   unsigned long N);
      template void register_host_memory<int                 , unsigned long>(int                 *& ptr,   unsigned long N);
      template void register_host_memory<unsigned int        , unsigned long>(unsigned int        *& ptr,   unsigned long N);
      template void register_host_memory<long                , unsigned long>(long                *& ptr,   unsigned long N);
      template void register_host_memory<unsigned long       , unsigned long>(unsigned long       *& ptr,   unsigned long N);
      template void register_host_memory<long long           , unsigned long>(long long           *& ptr, unsigned long N);
      template void register_host_memory<unsigned long long  , unsigned long>(unsigned long long  *& ptr, unsigned long N);
      template void register_host_memory<bool                , unsigned long>(bool                *& ptr, unsigned long N);
      template void register_host_memory<float               , unsigned long>(float               *& ptr, unsigned long N);
      template void register_host_memory<double              , unsigned long>(double              *& ptr, unsigned long N);
      template void register_host_memory<std::complex<float> , unsigned long>(std::complex<float> *& ptr, unsigned long N);
      template void register_host_memory<std::complex<double>, unsigned long>(std::complex<double>*& ptr, unsigned long N);


      template void register_host_memory<short               , unsigned long long>(short               *& ptr,   unsigned long long N);
      template void register_host_memory<unsigned short      , unsigned long long>(unsigned short      *& ptr,   unsigned long long N);
      template void register_host_memory<int                 , unsigned long long>(int                 *& ptr,   unsigned long long N);
      template void register_host_memory<unsigned int        , unsigned long long>(unsigned int        *& ptr,   unsigned long long N);
      template void register_host_memory<long                , unsigned long long>(long                *& ptr,   unsigned long long N);
      template void register_host_memory<unsigned long       , unsigned long long>(unsigned long       *& ptr,   unsigned long long N);
      template void register_host_memory<long long           , unsigned long long>(long long           *& ptr, unsigned long long N);
      template void register_host_memory<unsigned long long  , unsigned long long>(unsigned long long  *& ptr, unsigned long long N);
      template void register_host_memory<bool                , unsigned long long>(bool                *& ptr, unsigned long long N);
      template void register_host_memory<float               , unsigned long long>(float               *& ptr, unsigned long long N);
      template void register_host_memory<double              , unsigned long long>(double              *& ptr, unsigned long long N);
      template void register_host_memory<std::complex<float> , unsigned long long>(std::complex<float> *& ptr, unsigned long long N);
      template void register_host_memory<std::complex<double>, unsigned long long>(std::complex<double>*& ptr, unsigned long long N);

      // template instantiations      
      template void unregister_host_memory<int                  >(int                 *& ptr);
      template void unregister_host_memory<unsigned int         >(unsigned int        *& ptr);
      template void unregister_host_memory<long                 >(long                *& ptr);
      template void unregister_host_memory<unsigned long        >(unsigned long       *& ptr);
      template void unregister_host_memory<long long            >(long long           *& ptr);
      template void unregister_host_memory<unsigned long long   >(unsigned long long  *& ptr);
      template void unregister_host_memory<bool                 >(bool                *& ptr);
      template void unregister_host_memory<float                >(float               *& ptr);
      template void unregister_host_memory<double               >(double              *& ptr);
      template void unregister_host_memory<std::complex<float>  >(std::complex<float> *& ptr);
      template void unregister_host_memory<std::complex<double> >(std::complex<double>*& ptr);
    }

  }

}

#endif


