//-*-C++-*-

/***********************************************************
IBM Confidential
OCO Source Materials
IBM Research Zurich Knowledge Graph Licensed Internal Code
(C) Copyright IBM Corp. 2019
Authors: Peter W. J. Staar, Leonidas Georgopoulos,
         Christoph Auer, Aleksandros Sobczyk,
         Michele Dolfi, Dimitrios Christofidellis
Email: {taa, leg, cau, obc, dol, dic}@zurich.ibm.com
Date: 31/07/2019
**********************************************************/

#ifndef DEVICE_INITIALIZATION_CU
#define DEVICE_INITIALIZATION_CU

namespace func_lib
{
  namespace device_management
  {
    size_t get_global_memory()
    {

      // works for one device only
      int devCount;
      hipGetDeviceCount(&devCount);

      hipDeviceProp_t devProp;
      hipGetDeviceProperties(&devProp, devCount);

      return devProp.totalGlobalMem;

    }

    void get_device_report()
    {
      // Number of CUDA devices
      int devCount;
      hipGetDeviceCount(&devCount);

      std::cout << "CUDA Device Query...\n";
      std::cout << "There are %d CUDA devices.\n" << devCount << "\n";

      // Iterate through devices
      for(int i=0; i<devCount; ++i)
        {
          // Get device properties
          std::cout << "\n\n\n\tCUDA Device #" << i << "\n";

          hipDeviceProp_t devProp;
          hipGetDeviceProperties(&devProp, i);

          std::cout << "Major revision number:         " <<  devProp.major << "\n";
          std::cout << "Minor revision number:         " <<  devProp.minor << "\n";
          std::cout << "Name:                          " <<  devProp.name << "\n";
          std::cout << "Total global memory:           " <<  devProp.totalGlobalMem*(1.e-9) << " [giga-bytes]\n";
          std::cout << "Total shared memory per block: " <<  devProp.sharedMemPerBlock*(1.e-3) << " [kilo-bytes]\n";
          std::cout << "Total registers per block:     " <<  devProp.regsPerBlock << "\n";
          std::cout << "Warp size:                     " <<  devProp.warpSize << "\n";
          std::cout << "Maximum memory pitch:          " <<  devProp.memPitch << "\n";
          std::cout << "Maximum threads per block:     " <<  devProp.maxThreadsPerBlock << "\n";
          for (int i = 0; i < 3; ++i)
            {
              std::cout << "Maximum dimension %d of block:  " << i << devProp.maxThreadsDim[i] << "\n";
            }
          for (int i = 0; i < 3; ++i)
            {
              std::cout << "Maximum dimension %d of grid:   " << i << devProp.maxGridSize[i] << "\n";
            }
          std::cout << "Clock rate:                    " <<  devProp.clockRate << "\n";
          std::cout << "Total constant memory:         " <<  devProp.totalConstMem << "\n";
          std::cout << "Texture alignment:             " <<  devProp.textureAlignment << "\n";
          std::cout << "Concurrent copy and execution: " <<  (devProp.deviceOverlap ? "Yes" : "No") << "\n";
          std::cout << "Number of multiprocessors:     " <<  devProp.multiProcessorCount << "\n";
          std::cout << "Kernel execution timeout:      " <<  (devProp.kernelExecTimeoutEnabled ? "Yes" : "No") << "\n";

        }
    }

    int get_cuda_cores_per_sm(hipDeviceProp_t& devProp)
    {
      int cores = 0;
      std::stringstream error_msg;
      switch (devProp.major)
        {
        case 2: // Fermi
          if(devProp.minor==1)
            {
              cores=48;
            }
          else
            {
              cores=32;
            }
          break;
        case 3: // Kepler
          cores=192;
          break;
        case 5: // Maxwell
          cores=128;
          break;
        case 6: // Pascal
          if(devProp.minor==1)
            {
              cores=128;
            }
          else if(devProp.minor==0)
            {
              cores=64;
            }
          else
            {
              error_msg << __FILE__ << ":" << __LINE__ << ":" << __FUNCTION__ << " Unknown device type ";
              throw(error_msg);
            }
          break;
        case 7: // Volta
          if (devProp.minor==0)
            {
              cores=64;
            }
          else
            {
              error_msg << __FILE__ << ":" << __LINE__ << ":" << __FUNCTION__ << " Unknown device type ";
              throw(error_msg);
            }
          break;
        default:
          error_msg << __FILE__ << ":" << __LINE__ << ":" << __FUNCTION__ << " Unknown device type ";
          throw(error_msg);

        }
      return cores;
    }

    void cuda_check_for_error(const char *function,
                              const char *file,
                              const int   line,
                              const hipError_t error)
    {
      if ( hipSuccess != error )
        {
          std::stringstream error_msg;
          error_msg << file << ":" << line << ":" << function << "\t" << hipGetErrorString(error);
          throw(error_msg);
        }
    }

    bool cuda_check_for_error(const char *function,
                              const char *file,
                              const int   line )
    {
      //#ifdef CUDA_ERROR_CHECK

      hipError_t err = hipGetLastError();
      if ( hipSuccess != err )
        {
          fprintf( stderr, "cudaCheckError() failed in function %s \n\n\t\t --> file : %s \n\t\t --> line : %i \n\t\t --> error : %s\n", function, file, line, hipGetErrorString( err ) );
          exit( -1 );
        }

      // More careful checking. However, this will affect performance.
      // Comment away if needed.
      err = hipDeviceSynchronize();
      if( hipSuccess != err )
        {
          fprintf( stderr, "cudaCheckError() failed in function %s \n\n\t\t --> file : %s \n\t\t --> line : %i \n\t\t --> error : %s\n", function, file, line, hipGetErrorString( err ) );
          exit( -1 );
        }

      //#endif

      return true;
    }

    int get_total_number_of_devices()
    {
      int number=-1;
      hipGetDeviceCount(&number);
      return number;
    }

    int get_device_number()
    {
      int number=-1;
      hipError_t error = hipGetDevice(&number);
      cuda_check_for_error("get_device_number()", __FILE__, __LINE__, error);
      return number;
    }

    void set_device_number(int number)
    {
      hipError_t error = hipSetDevice(number);
      cuda_check_for_error("set_device_number()", __FILE__, __LINE__, error);
    }

    void synchronize_devices()
    {
      hipError_t error = hipDeviceSynchronize();
      cuda_check_for_error("synchronize_devices()", __FILE__, __LINE__, error);
    }

  }

}

#endif
