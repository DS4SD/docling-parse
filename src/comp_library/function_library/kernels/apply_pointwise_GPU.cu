#include "hip/hip_runtime.h"
//-*-C++-*-

#ifndef APPLY_POINTWISE_GPU_CU
#define APPLY_POINTWISE_GPU_CU

namespace func_lib
{

  namespace apply_pointwise_kernels
  {

    /* 
     *  INVERSE KERNEL
     */
    template<typename scalar_type, typename index_type>
    __global__ void kernel_inverse(index_type Nx, scalar_type* ptr, index_type N)
    {
      scalar_type ONE = 1;
      
      index_type ix = threadIdx.x;
      index_type Dx = blockDim.x;
      index_type Ix = blockIdx.x;

      index_type i = ix+Dx*Nx*Ix;
      for(index_type l=0; l<Nx; l++)
	{
	  if(i<N)
	    ptr[i] = ONE/ptr[i];

	  i += Dx;
	}
    }

    template<typename scalar_type, typename index_type>
    void execute_inverse(scalar_type* ptr, index_type N, int thread_id)
    {
      /*
      {
	std::stringstream ss;
	ss << "\t" << __FUNCTION__ << "\t id : " << thread_id << "\n";
	cout << ss.str();
      }
      */
      
      index_type Nx = 16;
      index_type Dx = 1024;
      
      index_type Bx = std::ceil((N+0.)/(Nx*Dx+0.));
      
      dim3 nb_threads(Dx);
      dim3 nb_blocks (Bx);

      hipStream_t stream = func_lib::device_management::get_stream_handle(thread_id, 0);

      kernel_inverse<<<nb_blocks, nb_threads, 0, stream>>>(Nx, ptr, N);
    }

    /* 
     *  SQUARE KERNEL
     */
    template<typename scalar_type, typename index_type>
    __global__ void kernel_square(index_type Nx, scalar_type* ptr, index_type N)
    {
      index_type ix = threadIdx.x;
      index_type Dx = blockDim.x;
      index_type Ix = blockIdx.x;

      index_type i = ix+Dx*Nx*Ix;
      for(index_type l=0; l<Nx; l++)
	{
	  if(i<N)
	    ptr[i] *= ptr[i];

	  i += Dx;
	}
    }

    template<typename scalar_type, typename index_type>
    void execute_square(scalar_type* ptr, index_type N, int thread_id)
    {
      /*
      {
	std::stringstream ss;
	ss << "\t" << __FUNCTION__ << "\t id : " << thread_id << "\t N : " << N << "\n";
	cout << ss.str();
      }
      */
      
      index_type Nx = 16;
      index_type Dx = 1024;
      
      index_type Bx = std::ceil((N+0.)/(Nx*Dx+0.));
      
      dim3 nb_threads(Dx);
      dim3 nb_blocks (Bx);

      hipStream_t stream = device_management::get_stream_handle(thread_id, 0);

      kernel_square<<<nb_blocks, nb_threads, 0, stream>>>(Nx, ptr, N);

      //device_management::cuda_check_for_error(__FUNCTION__, __FILE__, __LINE__);
    }

    /* 
     *  EXP KERNEL
     */
    template<typename scalar_type, typename index_type>
    __global__ void kernel_exp(index_type Nx, scalar_type* ptr, index_type N)
    {
      index_type ix = threadIdx.x;
      index_type Dx = blockDim.x;
      index_type Ix = blockIdx.x;

      index_type i = ix+Dx*Nx*Ix;
      for(index_type l=0; l<Nx; l++)
	{
	  if(i<N)
	    ptr[i] = exp(ptr[i]);

	  i += Dx;
	}
    }

    template<typename scalar_type, typename index_type>
    void execute_exp(scalar_type* ptr, index_type N, int thread_id)
    {
      index_type Nx = 16;
      index_type Dx = 1024;
      
      index_type Bx = std::ceil((N+0.)/(Nx*Dx+0.));
      
      dim3 nb_threads(Dx);
      dim3 nb_blocks (Bx);

      hipStream_t stream = func_lib::device_management::get_stream_handle(thread_id, 0);

      kernel_exp<<<nb_blocks, nb_threads, 0, stream>>>(Nx, ptr, N);
    }

    /* 
     *  SQRT KERNEL
     */
    template<typename scalar_type, typename index_type>
    __global__ void kernel_sqrt(index_type Nx, scalar_type* ptr, index_type N)
    {
      index_type ix = threadIdx.x;
      index_type Dx = blockDim.x;
      index_type Ix = blockIdx.x;

      index_type i = ix+Dx*Nx*Ix;
      for(index_type l=0; l<Nx; l++)
	{
	  if(i<N)
	    ptr[i] = sqrt(ptr[i]);

	  i += Dx;
	}
    }

    template<typename scalar_type, typename index_type>
    void execute_sqrt(scalar_type* ptr, index_type N, int thread_id)
    {
      index_type Nx = 16;
      index_type Dx = 1024;
      
      index_type Bx = std::ceil((N+0.)/(Nx*Dx+0.));
      
      dim3 nb_threads(Dx);
      dim3 nb_blocks (Bx);

      hipStream_t stream = func_lib::device_management::get_stream_handle(thread_id, 0);

      kernel_sqrt<<<nb_blocks, nb_threads, 0, stream>>>(Nx, ptr, N);
    }

    /* 
     *  SIGN KERNEL
     */
    template<typename scalar_type, typename index_type>
    __global__ void kernel_sign(index_type Nx, scalar_type* ptr, index_type N)
    {
      index_type ix = threadIdx.x;
      index_type Dx = blockDim.x;
      index_type Ix = blockIdx.x;

      index_type i = ix+Dx*Nx*Ix;
      for(index_type l=0; l<Nx; l++)
	{
	  if(i<N)
	    ptr[i] = ptr[i]>0? 1 : -1;

	  i += Dx;
	}
    }

    template<typename scalar_type, typename index_type>
    void execute_sign(scalar_type* ptr, index_type N, int thread_id)
    {
      index_type Nx = 16;
      index_type Dx = 1024;
      
      index_type Bx = std::ceil((N+0.)/(Nx*Dx+0.));
      
      dim3 nb_threads(Dx);
      dim3 nb_blocks (Bx);

      hipStream_t stream = func_lib::device_management::get_stream_handle(thread_id, 0);

      kernel_sign<<<nb_blocks, nb_threads, 0, stream>>>(Nx, ptr, N);
    }

    /* 
     *  CONSTANT KERNEL
     */
    template<typename scalar_type, typename index_type>
    __global__ void kernel_constant(index_type Nx, scalar_type* ptr, index_type N, scalar_type a)
    {
      index_type ix = threadIdx.x;
      index_type Dx = blockDim.x;
      index_type Ix = blockIdx.x;

      index_type i = ix+Dx*Nx*Ix;
      for(index_type l=0; l<Nx; l++)
	{
	  if(i<N)
	    ptr[i] = a;

	  i += Dx;
	}
    }


    template<typename scalar_type, typename index_type>
    void execute_constant(scalar_type* ptr, index_type N, scalar_type a, int thread_id)
    {
      index_type Nx = 16;
      index_type Dx = 1024;
      
      index_type Bx = std::ceil((N+0.)/(Nx*Dx+0.));
      
      dim3 nb_threads(Dx);
      dim3 nb_blocks (Bx);

      hipStream_t stream = func_lib::device_management::get_stream_handle(thread_id, 0);

      kernel_constant<<<nb_blocks, nb_threads, 0, stream>>>(Nx, ptr, N, a);
    }

    /* 
     *  KERNEL SCALE
     */
    template<typename scalar_type, typename index_type>
    __global__ void kernel_scale(index_type Nx, scalar_type* ptr, index_type N, scalar_type a)
    {
      index_type ix = threadIdx.x;
      index_type Dx = blockDim.x;
      index_type Ix = blockIdx.x;

      index_type i = ix+Dx*Nx*Ix;
      for(index_type l=0; l<Nx; l++)
	{
	  if(i<N)
	    ptr[i] *= a;

	  i += Dx;
	}
    }

    template<typename scalar_type, typename index_type>
    void execute_scale(scalar_type* ptr, index_type N, scalar_type a, int thread_id)
    {
      index_type Nx = 16;
      index_type Dx = 1024;
      
      index_type Bx = std::ceil((N+0.)/(Nx*Dx+0.));
      
      dim3 nb_threads(Dx);
      dim3 nb_blocks (Bx);

      hipStream_t stream = func_lib::device_management::get_stream_handle(thread_id, 0);

      kernel_scale<<<nb_blocks, nb_threads, 0, stream>>>(Nx, ptr, N, a);
    }    

    /* 
     *  KERNEL ADD
     */
    template<typename scalar_type, typename index_type>
    __global__ void kernel_add(index_type Nx, scalar_type* ptr, index_type N, scalar_type a)
    {
      index_type ix = threadIdx.x;
      index_type Dx = blockDim.x;
      index_type Ix = blockIdx.x;

      index_type i = ix+Dx*Nx*Ix;
      for(index_type l=0; l<Nx; l++)
	{
	  if(i<N)
	    ptr[i] += a;

	  i += Dx;
	}
    }

    template<typename scalar_type, typename index_type>
    void execute_add(scalar_type* ptr, index_type N, scalar_type a, int thread_id)
    {
      index_type Nx = 16;
      index_type Dx = 1024;
      
      index_type Bx = std::ceil((N+0.)/(Nx*Dx+0.));
      
      dim3 nb_threads(Dx);
      dim3 nb_blocks (Bx);

      hipStream_t stream = func_lib::device_management::get_stream_handle(thread_id, 0);

      kernel_add<<<nb_blocks, nb_threads, 0, stream>>>(Nx, ptr, N, a);
    }    

    
    // template instantiations

    /*
      template execute_exp    <scalar_type, index_type>(scalar_type* ptr, index_type N, int thread_id);
      template execute_sqrt   <scalar_type, index_type>(scalar_type* ptr, index_type N, int thread_id);
      template execute_square <scalar_type, index_type>(scalar_type* ptr, index_type N, int thread_id);
      template execute_inverse<scalar_type, index_type>(scalar_type* ptr, index_type N, int thread_id);

      template execute_constant<scalar_type, index_type>(scalar_type* ptr, index_type N, scalar_type a, int thread_id);
    */

    template void execute_exp    <float, int>(float* ptr, int N, int thread_id);
    template void execute_sqrt   <float, int>(float* ptr, int N, int thread_id);
    template void execute_sign   <float, int>(float* ptr, int N, int thread_id);
    template void execute_square <float, int>(float* ptr, int N, int thread_id);
    template void execute_inverse<float, int>(float* ptr, int N, int thread_id);

    template void execute_exp    <double, int>(double* ptr, int N, int thread_id);
    template void execute_sqrt   <double, int>(double* ptr, int N, int thread_id);
    template void execute_sign   <double, int>(double* ptr, int N, int thread_id);
    template void execute_square <double, int>(double* ptr, int N, int thread_id);
    template void execute_inverse<double, int>(double* ptr, int N, int thread_id);

    template void execute_exp    <float, unsigned long>(float* ptr, unsigned long N, int thread_id);
    template void execute_sqrt   <float, unsigned long>(float* ptr, unsigned long N, int thread_id);
    template void execute_sign   <float, unsigned long>(float* ptr, unsigned long N, int thread_id);
    template void execute_square <float, unsigned long>(float* ptr, unsigned long N, int thread_id);
    template void execute_inverse<float, unsigned long>(float* ptr, unsigned long N, int thread_id);

    template void execute_exp    <double, unsigned long>(double* ptr, unsigned long N, int thread_id);
    template void execute_sqrt   <double, unsigned long>(double* ptr, unsigned long N, int thread_id);
    template void execute_sign   <double, unsigned long>(double* ptr, unsigned long N, int thread_id);
    template void execute_square <double, unsigned long>(double* ptr, unsigned long N, int thread_id);
    template void execute_inverse<double, unsigned long>(double* ptr, unsigned long N, int thread_id);

    template void execute_constant<float , int>(float* ptr , int N   , float a, int thread_id);
    template void execute_constant<double, int>(double* ptr, int N   , double a, int thread_id);

    template void execute_constant<float , unsigned long>(float* ptr , unsigned long N, float a, int thread_id);
    template void execute_constant<double, unsigned long>(double* ptr, unsigned long N, double a, int thread_id);

    template void execute_scale<float , int>(float* ptr , int N   , float a, int thread_id);
    template void execute_scale<double, int>(double* ptr, int N   , double a, int thread_id);

    template void execute_scale<float , unsigned long>(float* ptr , unsigned long N, float a, int thread_id);
    template void execute_scale<double, unsigned long>(double* ptr, unsigned long N, double a, int thread_id);

    template void execute_add<float , int>(float* ptr , int N   , float a, int thread_id);
    template void execute_add<double, int>(double* ptr, int N   , double a, int thread_id);

    template void execute_add<float , unsigned long>(float* ptr , unsigned long N, float a, int thread_id);
    template void execute_add<double, unsigned long>(double* ptr, unsigned long N, double a, int thread_id);
  }

}

#endif
